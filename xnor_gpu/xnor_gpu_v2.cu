#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <sstream>
#include <utility>
#include <unordered_map>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <chrono>
#include <vector>
#include <assert.h>
#include <math.h>

#define NUM_STREAMS 2

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

// This is second version of the gpu implementation
// This version a general benchmarking to compare with CPU,
// Binary operations will be handled single convolution kernel to utilize register memory usage
constexpr std::pair<int, int> register_size(8, 4);
constexpr int nTPB=256;

template <typename T>
struct matrix1d {
	int lenght;
	T *arr;
};

template <typename T>
struct matrix2d {
	int row;
	int col;
	T *arr;
};

template <typename T>
struct matrix3d {
	int row;
	int col;
	int channel;
	T *arr;
};

template <typename T>
struct matrix4d{
	int row;
	int col;
	int channel_in;
	int channel_out;
	T *arr;
};


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


std::pair<int, int> find_binary_size(std::pair<int, int>input_size,  std::pair<int, int>kernel_size){
	int size_x = ceil((input_size.first - register_size.first)
						/static_cast<double>(register_size.first + 1 - kernel_size.first) + 1);
	int size_y = ceil((input_size.second - register_size.second )
						/static_cast<double>(register_size.second + 1 - kernel_size.second) + 1);
	if (size_x < 0)
		size_x = 1;
	if (size_y < 0)
		size_y = 1;
	return std::make_pair(size_x, size_y);
}

size_t choose_block_size(size_t val){
  if (val >= nTPB) return nTPB;
  if (val <= 32) return 32;
  val = (val >> 1) | val;
  val = (val >> 2) | val;
  val = (val >> 4) | val;
  val = (val >> 8) | val;
  val = (val >> 16) | val;
  val++;
  return val;
}

void int2binary(float* input_x, const std::pair<int, int> input_index,
 std::pair<int, int> output_location, unsigned int &output_y, const std::pair<int ,int>register_size, int input_col)
{
	int sign = 0;
	long int pozitive = 1;
	long int negative = 0;
	int count = output_location.second * register_size.second  + output_location.first;

	assert(count < register_size.second * register_size.first);

	for (int j=0; j<register_size.second; j++)
	{
		for(int i=0; i<register_size.first; i++)
		{
			sign = (input_x[(input_index.second) * input_col+ input_index.first + i] > 0) - (input_x[(input_index.second) * input_col+ input_index.first + i] < 0);
			if (sign == 1)
			{
				output_y = pozitive<<count | output_y;
			}
			else if (sign == -1)
			{
				output_y = negative<<count | output_y;
			}
			else
			{
				output_y = negative<<count |output_y;
			}
			if ((input_index.first + i) >=  input_col)
			{
				break;
			}
			count++;
		}
	}

}

void intMat2BinaryMat(float *const& input_mat, unsigned int *const& binary_mat, std::pair<int, int> kernel_size, int input_row, int input_col, int binary_col, int binary_row)
{
	//float * input_mat = input_tensor.arr[i * input_tensor.channel_in + j];
	//unsigned int * binary_mat = binary_tensor.arr[i * input_tensor.channel_in + j];
	int index_x = 0;
	int index_y = 0;
	std::pair<int, int> input_index(0, 0);
	std::pair<int, int> output_location(0, 0);

	// Test
	while(input_row >= input_index.second)
	{
		int i = 0;
		input_index.first = 0;
		index_x = 0;

		while(input_col > i)
		{
			i = input_index.first + register_size.first;
			int2binary(input_mat, input_index, output_location, binary_mat[index_y *binary_col + index_x], register_size, input_col);
			input_index.first = input_index.first + register_size.first + 1 - kernel_size.first;
			index_x++;

		}
		output_location.second++;
		input_index.second++;
		if(input_index.second >= input_row)
			{
				break;
			}
		if (output_location.second % register_size.second == 0)
		{
			output_location.second = 0;
			input_index.second = input_index.second + 1 - kernel_size.second;
			index_y++;
		}
	}
}
std::pair<int, int> BinaryMatMemoryAllocation( std::pair<int, int> input_size, std::pair<int, int> kernel_size)
{
	int size_x = ceil((input_size.first - register_size.first)
						/static_cast<double>(register_size.first + 1 - kernel_size.first) + 1);
	int size_y = ceil((input_size.second - register_size.second )
						/static_cast<double>(register_size.second + 1 - kernel_size.second) + 1);
	if (size_x < 0)
		size_x = 1;
	if (size_y < 0)
		size_y = 1;

	return std::make_pair(size_x, size_y);
}
template <typename T>
__global__ void compK_matrix(T* input_data, T kernel_value,
    T* output_data, int channel_in, int width, int height) {

    float accum;
    int col = threadIdx.x + blockIdx.x * blockDim.x;   //col index
    int row = threadIdx.y + blockIdx.y * blockDim.y;   //row index
    int mask_row_radius = mask_rows / 2;
    int mask_col_radius = mask_cols / 2;


    for (int k = 0; k < channel_in; k++) {      
        if (row < height && col < width) {
            accum = 0;
            int start_row = row - mask_row_radius;  
            int start_col = col - mask_col_radius;  

            for (int i = 0; i < mask_rows; i++) { 

                for (int j = 0; j < mask_cols; j++) { 

                    int row_index = start_row + i; 
                    int col_index = start_col + j; 

                    if (row_index >= 0 && row_index < height && col_index >= 0 && col_index < width) {

                        accum += input_data[(row_index * width + col_index) * channel_in + k] *
                            kernel_value;
                    }
                    else accum += 0;
                }

            }
            output_data[(row * width + col) * channel_in + k] = accum;
        }

    }
}

void __global__ zeroPadding(float* input_tensor, float* output_tensor,  int kernel_row, int kernel_col, int input_col, int input_row, int output_col, int output_row, int output_channel)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int op_buffer = idx / output_col; // simple buffer for same operation
	int index_x = (idx % output_col) - (kernel_col - 1)/ 2;
	int index_y = op_buffer%output_row - (kernel_row - 1)/ 2;
	int index_z = op_buffer / output_row;
	if (idx< output_row * output_col * output_channel)
	{
		if(index_x >= 0 && index_y >= 0 )
		{
			if( index_x < input_col && index_y < input_row )
			{
				output_tensor[idx] = input_tensor[(index_z * input_col * input_row ) + ( index_y * input_col ) + index_x];
			}
		}
		else {
			output_tensor[idx] = 0;
		}
	}
}

void __global__ kernel_sum(
		const unsigned int *   d_idata,
		float *  d_odata,
        const int col,
        const int row,
        const int channel_in,
        const int channel_out)
{
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx < (col * row * channel_out))
	{

		int tidx = idx%(col*row) + ((idx/(col*row) ) *(col * row * channel_in) ); // indexing for 4 dim , since kernel must sum values with same channel out
		int tsum = 0;
		#pragma unroll
		for (int i = 0; i < channel_in; i++)
		{
			tsum += d_idata[tidx];
			tidx += row * col;
		}
		d_odata[idx] = static_cast<float>(tsum);// / static_cast<float>(channel_in);
	}
}

template<typename T>
__device__ void to_binary_register(
	const T &idata,
	unsigned int &odata,
	 int *output_location)
{
	int sign = (idata > 0) - (idata < 0);
	const unsigned int pozitive = 1;
	const unsigned int negative = 0;
	//int count = output_location[1] * register_size.second  + output_location[0];
	//assert(count < register_size.second * register_size.first);
	if (sign > -1)
	{
		odata = pozitive<<(output_location[1] * register_size.first  + output_location[0]) | odata;
	}
	else
	{
		odata = negative<<(output_location[1] * register_size.first  + output_location[0]) | odata;
	}
}

template<typename T>
void __global__  convert2binary(
	const T *  d_idata,
	unsigned int *  d_odata,
	const int row, const int b_row,
	const int col, const int b_col,
	const int channel,
	const int kernel_row = 3, const int kernel_col = 3)
{
	// Each thread will store a size = 32 array inside their single register
	int idx = threadIdx.x+blockDim.x*blockIdx.x; //register IDX
	// n*(regsiter_size - kernel_size)
	if (idx < (b_row * b_col * channel))
	{

		int input_index[] = {(idx%b_col) * (register_size.first - kernel_col), ((idx/b_col) % b_row)* (register_size.second - kernel_row), (idx/(b_col * b_row) )}; // x, y ,z
		int data_idx = input_index[0] + (input_index[1] * col) + (input_index[2] * row * col);
		//int input_index[] = {data_idx%row, data_idx/col, data_idx/(row*col)}; // from start of array , (x, y, z)
		int register_location[] = {0, 0};
		unsigned int local_register = 0;
		for (int j=0; register_size.second>j; j++)
		{
			for (int i=0; register_size.first>i; i++)
			{
				to_binary_register<T>(d_idata[data_idx], local_register, register_location);
				++data_idx;
				input_index[0] += 1;
				register_location[0] += 1;
				if (input_index[0] == col) break;
			}
			data_idx = data_idx + col - register_location[0];
			input_index[1] += 1;
			input_index[0] = (idx%b_col) * (register_size.first - kernel_col);
			register_location[0] = 0;
			register_location[1] += 1;
			if (input_index[1] == row) break;
		}
		d_odata[idx] = local_register;
	}
}
template<typename T>
void __global__ scalar_multiplication(T* __restrict__ d_idata, const T __restrict__ scalar, const int height, const int width)
{
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx<height * width)
	{
		d_idata[idx] = d_idata[idx] * scalar;
	}
}


void __global__ scaling_result(T* __restrict__ d_idata, const T* __restrict__ d_scalar, const int height, const int width, const int channel_out)
{
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
	if (idx<height * width * channel_out)
	{
		d_idata[idx] = d_idata[idx] * d_scalar[idx%(height * width)];
	}
}

void __device__ binary2int(const unsigned int  idata,  unsigned int &odata, int kernel_row, int kernel_col)
{
	constexpr unsigned int mask = 1;
	unsigned int shifter = 0;
	unsigned int buffer = 0;
	for (int j=0; kernel_row>j; ++j)
	{
		for(int i=0; kernel_col>i; ++i)
		{
			buffer += (idata >> shifter) & mask;
			++shifter;
		}
		shifter += register_size.first - kernel_col;
	}
	odata = 2 * buffer - (kernel_row * kernel_col);
}


void __global__ binaryConv2d(
		const unsigned int * input_tensor,
		unsigned int * output_tensor,
		const unsigned int * weight_tensor,
		int input_row, int input_col,
		int kernel_row, int kernel_col,
		int output_row, int output_col,
		int channel_in, int channel_out
		)
{

	int idx = threadIdx.x +blockDim.x*blockIdx.x;
	int conv_per_row = register_size.second - (kernel_row - 1);
	int conv_per_column = register_size.first - (kernel_col - 1);
	int output_index_x = (idx % input_col) * conv_per_column;
	int output_index_y = ((idx / input_col) % input_row) * conv_per_row;

	if (idx < input_row * input_col * channel_in * channel_out)
	{
		unsigned int register_buffer = input_tensor[idx % (input_row * input_col * channel_in)];
		if ( (output_index_x + conv_per_column) > output_col)
		{
			conv_per_column = output_col - output_index_x;
		}
		if ( (output_index_y + conv_per_row) > output_row)
		{
			conv_per_row = output_row - output_index_y;
		}

		unsigned int mask = std::pow(2, kernel_col) - 1;
		for (int j=1; kernel_row > j; j++)
		{
			mask = (mask<<register_size.first) | static_cast<unsigned int>(std::pow(2, kernel_col) - 1);
		}
		int default_index = (idx / (input_row * input_col) ) *  (output_col * output_row);
		auto weight_index = idx / (input_row * input_col);
		unsigned int shifter = 0;
		for (int j=0; conv_per_row>j; ++j)
		{
			for (int i=0; conv_per_column>i; ++i)
			{
				unsigned int buffer = (~(register_buffer>>shifter) ^ (weight_tensor[weight_index]) ) & mask;
				binary2int(buffer, output_tensor[default_index + (output_index_y+j)*output_col + output_index_x + i], kernel_row, kernel_col);
				++shifter;
			}
			// Check if register is not fully filled,
			// if not add shifter the missing shift amount
			shifter += register_size.second - conv_per_column;
		}
	}

}





// This part must be updated to concurrent execution
void xnor_convolution(matrix3d<float> &h_input_tensor, matrix4d<unsigned int> &h_weight_tensor, matrix3d<float> &h_output_tensor, const float alpha, int kernel_row, int kernel_col, bool padding=true)
{

	hipEvent_t start, stop;
	hipEvent_t start1, stop1;
	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);


	matrix3d<float> d_input_tensor;
	d_input_tensor.col = h_input_tensor.col;
	d_input_tensor.row = h_input_tensor.row;
	d_input_tensor.channel = h_input_tensor.channel;
	auto copy_size = sizeof(float) * d_input_tensor.col* d_input_tensor.row * d_input_tensor.channel;
	hipMalloc((void **)&d_input_tensor.arr, copy_size);
	hipMemcpy(d_input_tensor.arr, h_input_tensor.arr, copy_size, hipMemcpyHostToDevice);
	//
	// Calculate K matrix
	// Use async steam2
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	matrix2d<float> d_K_matrix;
	d_K_matrix.col = h_input_tensor.col;
	d_K_matrix.row = h_input_tensor.row;
	copy_size = sizeof(float) * d_K_matrix.col* d_K_matrix.row;
	hipMalloc((void **)&d_K_matrix.arr, copy_size);
	const float kernel_value = 1.0 / static_cast<float>(h_weight_tensor.row * h_weight_tensor.col);
	auto block_size = choose_block_size(h_input_tensor.row * h_input_tensor.col);
	auto grid_size = (h_input_tensor.row * h_input_tensor.col+ block_size - 1)/block_size; 
	compK_matrix<float><<<grid_size, block_size, stream1>>>(d_input_tensor.arr, kernel_value,
		d_K_matrix.arr, d_input_tensor.channel, d_input_tensor.width, d_input_tensor.height);
	//
	scalar_multiplication<float><<<grid_size, block_size, stream1>>>(d_K_matrix.arr, alpha, height, width);
	matrix3d<float> d_padded_input_tensor;
	d_padded_input_tensor.row = h_input_tensor.row + kernel_row - 1;
	d_padded_input_tensor.col = h_input_tensor.col + kernel_col - 1;
	d_padded_input_tensor.channel = h_input_tensor.channel;
	copy_size = sizeof(float) * d_padded_input_tensor.row * d_padded_input_tensor.col * d_padded_input_tensor.channel;
	gpuErrchk(hipMalloc((void **)&d_padded_input_tensor.arr, copy_size));

	block_size = choose_block_size(d_padded_input_tensor.row * d_padded_input_tensor.col * d_padded_input_tensor.channel);
	grid_size = (d_padded_input_tensor.row * d_padded_input_tensor.col * d_padded_input_tensor.channel + block_size - 1)/block_size;
	zeroPadding<<<grid_size, block_size>>>(d_input_tensor.arr, d_padded_input_tensor.arr,  kernel_row, kernel_col, d_input_tensor.col, d_input_tensor.row, d_padded_input_tensor.row, d_padded_input_tensor.col, d_padded_input_tensor.channel);
	//hipFree(d_input_tensor.arr);
	auto binary_size = find_binary_size(std::make_pair(h_input_tensor.col, h_input_tensor.row), std::make_pair(kernel_col, kernel_row));

	matrix3d<unsigned int> d_binary_input_tensor;
	d_binary_input_tensor.row = binary_size.second;
	d_binary_input_tensor.col = binary_size.first;
	d_binary_input_tensor.channel = d_padded_input_tensor.channel;
	copy_size = sizeof(unsigned int) * d_binary_input_tensor.row * d_binary_input_tensor.col * d_binary_input_tensor.channel;

	gpuErrchk(hipMalloc((void **)&d_binary_input_tensor.arr, copy_size));
	hipEventRecord(start, 0);
	convert2binary<<<grid_size, block_size>>>(d_padded_input_tensor.arr, d_binary_input_tensor.arr,
			d_padded_input_tensor.row, d_binary_input_tensor.row,
			d_padded_input_tensor.col, d_binary_input_tensor.col,
			d_binary_input_tensor.channel,
			kernel_row, kernel_col);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	std::cout<<"Int2Binary Time= "<< milliseconds<<std::endl;
	//hipFree(d_padded_input_tensor.arr);
	matrix4d<unsigned int> d_convolution_buffer;
	d_convolution_buffer.col = h_input_tensor.col;
	d_convolution_buffer.row = h_input_tensor.row;
	d_convolution_buffer.channel_in = h_input_tensor.channel;
	d_convolution_buffer.channel_out = h_weight_tensor.channel_out;
	copy_size = sizeof(unsigned int) * d_convolution_buffer.col * d_convolution_buffer.row * d_convolution_buffer.channel_in * d_convolution_buffer.channel_out;
	gpuErrchk(hipMalloc((void **)& d_convolution_buffer.arr, copy_size));
	matrix4d<unsigned int> d_weight_tensor;
	d_weight_tensor.row = h_weight_tensor.row;
	d_weight_tensor.col = h_weight_tensor.col;
	d_weight_tensor.channel_in = h_weight_tensor.channel_in;
	d_weight_tensor.channel_out = h_weight_tensor.channel_out;
	copy_size = sizeof(unsigned int) * d_weight_tensor.row *d_weight_tensor.col * d_weight_tensor.channel_in * d_weight_tensor.channel_out;
	gpuErrchk(hipMalloc((void**)&d_weight_tensor.arr, copy_size)); // pinned memory can be tested
	hipMemcpy(d_weight_tensor.arr, h_weight_tensor.arr, copy_size, hipMemcpyHostToDevice);
	block_size = choose_block_size(d_convolution_buffer.col * d_convolution_buffer.row * d_convolution_buffer.channel_in * d_convolution_buffer.channel_out);
	grid_size = (d_convolution_buffer.col* d_convolution_buffer.row * d_convolution_buffer.channel_in * d_convolution_buffer.channel_out + block_size - 1)/ block_size;
	hipEventRecord(start1, 0);
	binaryConv2d<<<grid_size, block_size>>>(d_binary_input_tensor.arr, d_convolution_buffer.arr, d_weight_tensor.arr
			,d_binary_input_tensor.row, d_binary_input_tensor.col
			, kernel_row, kernel_col
			,d_convolution_buffer.row, d_convolution_buffer.col
			,d_convolution_buffer.channel_in, d_convolution_buffer.channel_out
			);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&milliseconds, start1, stop1);
	std::cout<<"Convolution Time= "<< milliseconds<<std::endl;
	hipFree(d_binary_input_tensor.arr);
	matrix3d<float> d_output_tensor;
	d_output_tensor.col = h_output_tensor.col;
	d_output_tensor.row = h_output_tensor.row;
	d_output_tensor.channel = h_output_tensor.channel;
	copy_size = sizeof(float) * d_output_tensor.row * d_output_tensor.col * d_output_tensor.channel;
	hipMalloc((void**)&d_output_tensor.arr, copy_size);
	block_size = choose_block_size(d_output_tensor.row * d_output_tensor.col * d_output_tensor.channel);
	grid_size = (d_output_tensor.row * d_output_tensor.col * d_output_tensor.channel + block_size - 1) / block_size;
	hipEventRecord(start2, 0);
	kernel_sum<<<grid_size, block_size>>>(d_convolution_buffer.arr, d_output_tensor.arr, d_output_tensor.col, d_output_tensor.row, d_convolution_buffer.channel_in, d_convolution_buffer.channel_out);
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&milliseconds, start2, stop2);
	std::cout<<"Summation Time= "<< milliseconds<<std::endl;
	hipDeviceSynchronize()
	hipStreamDestroy(stream1);
	// Multiplication with K and alpha
	//scaling_result<<<>>>();
	//hipFree(d_convolution_buffer.arr);
	hipMemcpy(h_output_tensor.arr, d_output_tensor.arr, copy_size, hipMemcpyDeviceToHost);
	//hipFree(d_output_tensor.arr);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	hipEventDestroy(start2);
	hipEventDestroy(stop2);

	return;

}



int main()
{
	int row = 512;
	int col = 512;
	int kernel_row = 3;
	int kernel_col = 3;

	int channel_in = 1;
	int channel_out = 1;
	matrix3d<float> input_tensor;
	matrix4d<float> weight_tensor;
	input_tensor.row = row;
	input_tensor.col = col;
	input_tensor.channel = channel_in;
	// Init Matrices
	input_tensor.arr = new float [input_tensor.channel * input_tensor.row * input_tensor.col];
	weight_tensor.row = kernel_row;
	weight_tensor.col = kernel_col;
	weight_tensor.channel_in = channel_in;
	weight_tensor.channel_out = channel_out;
	weight_tensor.arr = new float [weight_tensor.channel_in * weight_tensor.channel_out * weight_tensor.row * weight_tensor.col];

	bool padding = true;
	// Default Values
	for(int i=0; input_tensor.channel > i; ++i)
	{
		for (int j=0; input_tensor.col * input_tensor.row> j; ++j)
		{
			input_tensor.arr[i * input_tensor.col * input_tensor.row + j] = (rand() % 50) - 25;
		}
	}
	for(int i=0; weight_tensor.channel_in * weight_tensor.channel_out > i; ++i)
	{
		for (int j=0; weight_tensor.col * weight_tensor.row> j; ++j)
		{
			weight_tensor.arr[i * weight_tensor.col * weight_tensor.row + j] = (rand() % 50) -25;
		}
	}
	// Make Weights binary as preProcessing
	auto weight_size = BinaryMatMemoryAllocation(std::make_pair(weight_tensor.row, weight_tensor.col), std::make_pair(weight_tensor.col, weight_tensor.row));
	matrix4d<unsigned int> binary_weight_tensor;
	binary_weight_tensor.col = weight_size.first;
	binary_weight_tensor.row = weight_size.second;
	binary_weight_tensor.channel_in = weight_tensor.channel_in;
	binary_weight_tensor.channel_out = weight_tensor.channel_out;
	binary_weight_tensor.arr = new unsigned int [binary_weight_tensor.channel_in * binary_weight_tensor.channel_out *binary_weight_tensor.row * binary_weight_tensor.col];
	for (int i= 0; weight_tensor.channel_out > i; ++i)
	{
		for(int j=0; weight_tensor.channel_in > j; ++j)
		{
			intMat2BinaryMat(&weight_tensor.arr[(i * weight_tensor.channel_in + j) * weight_tensor.row * weight_tensor.col], &binary_weight_tensor.arr[i * weight_tensor.channel_in + j],
					std::make_pair(weight_tensor.col, weight_tensor.row), weight_tensor.row, weight_tensor.col, binary_weight_tensor.col, binary_weight_tensor.row);
		}
	}
	delete weight_tensor.arr;
	// A sample layer
	matrix3d<float> output_tensor;
	output_tensor.col = input_tensor.col;
	output_tensor.row = input_tensor.row;
	output_tensor.channel = input_tensor.channel;
	output_tensor.arr = new float [input_tensor.col* input_tensor.row * input_tensor.channel];
	xnor_convolution(input_tensor, binary_weight_tensor, output_tensor, weight_tensor.row, weight_tensor.col ,padding);

	delete[] input_tensor.arr;
	delete[] binary_weight_tensor.arr;
	delete[] output_tensor.arr;
	return 0;
}



